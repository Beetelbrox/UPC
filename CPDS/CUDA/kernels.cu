#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>

__global__ void gpu_Heat (float *h, float *g, int N, float *residual) {
    extern __shared__ float v_reduction[]; // Vector to store the reduction values
    int block_id = (blockIdx.x + blockIdx.y*gridDim.x); 
    int t_id = block_id*blockDim.x*blockDim.y + threadIdx.x + threadIdx.y*blockDim.x;    
    int red_ix = threadIdx.x*blockDim.x + threadIdx.y;
    v_reduction[red_ix];
    // As we have created thread in excess, only proceed if the id of the thread is within the size of the matrix of elements to process.
    if(t_id < (N-2)*(N-2)){
      // As the area we have to process is smaller than u/uhelp and we have created threads in excess, let's do a
      // transformation to obtain the index of the position in u/uhelp tha the thread has to modify, so we can have cleaner code later on.
      int mod_id = N               // First Row of full matrix
                 + (t_id/(N-2))*N  // y-coord in small matrix times full row matrix
                 + 1               // First element of the row in the full matrix
                 + (t_id%(N-2));   // x-coord in small matrix
    
      g[mod_id] = 0.25*( h[mod_id-N]  // Up
                       + h[mod_id+1]  // Right
                       + h[mod_id+N]  // Down 
                       + h[mod_id-1]  // Left
                       );
      float diff = g[mod_id]-h[mod_id];
      v_reduction[red_ix] = diff*diff;
      
      __syncthreads();
    }
    
    // Reduction
    for(int s = (blockDim.x*blockDim.x)>>1; s>0; s>>=1){
      if(red_ix < s) v_reduction[red_ix] += v_reduction[red_ix+s];
      __syncthreads();
    }
    if(red_ix == 0) residual[blockIdx.x*gridDim.x+blockIdx.y] = v_reduction[0];
}
